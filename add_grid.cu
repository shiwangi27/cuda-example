#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>

__global__
void add(int n, float *a, float *b) {
    index = blockIdx.x * blockDim.x + threadIdx.x;
    stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride) {
        b[i] = a[i] + b[i]; 
    }
}

int main(void) {
    int N = 1 << 20;
    
    float *a, *b;
    hipMallocManaged(&a, N * sizeof(float));
    hipMallocManaged(&b, N * sizeof(float));

    float *a = new float[N];
    float *b = new float[N];

    for (int i=0; i < N; i++){
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    add<<<1, 256>>>(N, a, b);

    hipDeviceSynchronize();

    float maxError = 0.0f;
    for (int i=0; i<N; i++) {
        maxError = fmax(maxError, fabs(b[i]-3.0f));
    }
    std::cout << "Max error: " << maxError << std::endl;

    hipFree(a); 
    hipFree(b);

    return 0;
}

